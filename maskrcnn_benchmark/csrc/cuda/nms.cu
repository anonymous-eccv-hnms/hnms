#include "hip/hip_runtime.h"
// Copyright (c) Facebook, Inc. and its affiliates. All Rights Reserved.
#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>

#include <THC/THC.h>
#include <THC/THCDeviceUtils.cuh>

#include <vector>
#include <iostream>

#include <ctime>
#include <chrono>
using namespace std::chrono;

#define CONF_TO_INT_MULT 1000000
#define CONF_TO_INT_ADD 100000
#define CONF_TO_INT(x) (long long)((x) * CONF_TO_INT_MULT) + CONF_TO_INT_ADD

#define CUDA_1D_KERNEL_LOOP(i, n)                            \
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < n; \
       i += blockDim.x * gridDim.x)
const int CUDA_NUM_THREADS = 512;

int const threadsPerBlock = sizeof(unsigned long long) * 8;

__device__ inline float devIoU(float const * const a, float const * const b) {
  float left = max(a[0], b[0]), right = min(a[2], b[2]);
  float top = max(a[1], b[1]), bottom = min(a[3], b[3]);
  float width = max(right - left + 1, 0.f), height = max(bottom - top + 1, 0.f);
  float interS = width * height;
  float Sa = (a[2] - a[0] + 1) * (a[3] - a[1] + 1);
  float Sb = (b[2] - b[0] + 1) * (b[3] - b[1] + 1);
  return interS / (Sa + Sb - interS);
}

__global__ void nms_kernel(const int n_boxes, const float nms_overlap_thresh,
                           const float *dev_boxes, unsigned long long *dev_mask) {
  const int row_start = blockIdx.y;
  const int col_start = blockIdx.x;

  // if (row_start > col_start) return;

  const int row_size =
        min(n_boxes - row_start * threadsPerBlock, threadsPerBlock);
  const int col_size =
        min(n_boxes - col_start * threadsPerBlock, threadsPerBlock);

  __shared__ float block_boxes[threadsPerBlock * 5];
  if (threadIdx.x < col_size) {
    block_boxes[threadIdx.x * 5 + 0] =
        dev_boxes[(threadsPerBlock * col_start + threadIdx.x) * 5 + 0];
    block_boxes[threadIdx.x * 5 + 1] =
        dev_boxes[(threadsPerBlock * col_start + threadIdx.x) * 5 + 1];
    block_boxes[threadIdx.x * 5 + 2] =
        dev_boxes[(threadsPerBlock * col_start + threadIdx.x) * 5 + 2];
    block_boxes[threadIdx.x * 5 + 3] =
        dev_boxes[(threadsPerBlock * col_start + threadIdx.x) * 5 + 3];
    block_boxes[threadIdx.x * 5 + 4] =
        dev_boxes[(threadsPerBlock * col_start + threadIdx.x) * 5 + 4];
  }
  __syncthreads();

  if (threadIdx.x < row_size) {
    const int cur_box_idx = threadsPerBlock * row_start + threadIdx.x;
    const float *cur_box = dev_boxes + cur_box_idx * 5;
    int i = 0;
    unsigned long long t = 0;
    int start = 0;
    if (row_start == col_start) {
      start = threadIdx.x + 1;
    }
    for (i = start; i < col_size; i++) {
      if (devIoU(cur_box, block_boxes + i * 5) > nms_overlap_thresh) {
        t |= 1ULL << i;
      }
    }
    const int col_blocks = THCCeilDiv(n_boxes, threadsPerBlock);
    dev_mask[cur_box_idx * col_blocks + col_start] = t;
  }
}

// CUDA: number of blocks for threads.
inline int GET_BLOCKS(const int N) {
  return (N + CUDA_NUM_THREADS - 1) / CUDA_NUM_THREADS;
}


template <typename T>
__global__ void hnms_max_conf_kernel(long long nthreads,
        T* box_confs,
        int64_t* cell_indices,
        int64_t* cell_max_confs) {
  CUDA_1D_KERNEL_LOOP(i, nthreads) {
      unsigned long long conf = CONF_TO_INT(box_confs[i]);
      unsigned long long cell = cell_indices[i];
      unsigned long long * cell_max = (unsigned long long*)(cell_max_confs + cell);
      // long long type is not supported for atomiMax
      atomicMax(cell_max, conf);
  }
}

template <typename T>
__global__ void hnms_max_idx_kernel(long long nthreads,
        T* box_confs,
        int64_t* cell_indices,
        int64_t* cell_max_confs) {
  CUDA_1D_KERNEL_LOOP(i, nthreads) {
      unsigned long long conf = CONF_TO_INT(box_confs[i]);
      auto cell = cell_indices[i];
      unsigned long long* cell_max = (unsigned long long*)(cell_max_confs + cell);
      // no implementation to take long long, but unsigned long long
      atomicCAS(cell_max, conf, (unsigned long long)i);
  }
}

// boxes is a N x 5 tensor
at::Tensor nms_cuda(const at::Tensor boxes, float nms_overlap_thresh) {
  using scalar_t = float;
  AT_ASSERTM(boxes.type().is_cuda(), "boxes must be a CUDA tensor");
  auto scores = boxes.select(1, 4);
  auto order_t = std::get<1>(scores.sort(0, /* descending=*/true));
  auto boxes_sorted = boxes.index_select(0, order_t);

  int boxes_num = boxes.size(0);

  const int col_blocks = THCCeilDiv(boxes_num, threadsPerBlock);

  scalar_t* boxes_dev = boxes_sorted.data<scalar_t>();

  THCState *state = at::globalContext().lazyInitCUDA(); // TODO replace with getTHCState

  unsigned long long* mask_dev = NULL;
  //THCudaCheck(THCudaMalloc(state, (void**) &mask_dev,
  //                      boxes_num * col_blocks * sizeof(unsigned long long)));

  mask_dev = (unsigned long long*) THCudaMalloc(state, boxes_num * col_blocks * sizeof(unsigned long long));

  dim3 blocks(THCCeilDiv(boxes_num, threadsPerBlock),
              THCCeilDiv(boxes_num, threadsPerBlock));
  dim3 threads(threadsPerBlock);
  nms_kernel<<<blocks, threads>>>(boxes_num,
                                  nms_overlap_thresh,
                                  boxes_dev,
                                  mask_dev);

  std::vector<unsigned long long> mask_host(boxes_num * col_blocks);
  THCudaCheck(hipMemcpy(&mask_host[0],
                        mask_dev,
                        sizeof(unsigned long long) * boxes_num * col_blocks,
                        hipMemcpyDeviceToHost));

  std::vector<unsigned long long> remv(col_blocks);
  memset(&remv[0], 0, sizeof(unsigned long long) * col_blocks);

  at::Tensor keep = at::empty({boxes_num}, boxes.options().dtype(at::kLong).device(at::kCPU));
  int64_t* keep_out = keep.data<int64_t>();

  int num_to_keep = 0;
  for (int i = 0; i < boxes_num; i++) {
    int nblock = i / threadsPerBlock;
    int inblock = i % threadsPerBlock;

    if (!(remv[nblock] & (1ULL << inblock))) {
      keep_out[num_to_keep++] = i;
      unsigned long long *p = &mask_host[0] + i * col_blocks;
      for (int j = nblock; j < col_blocks; j++) {
        remv[j] |= p[j];
      }
    }
  }

  THCudaFree(state, mask_dev);
  // TODO improve this part
  return std::get<0>(order_t.index({
                       keep.narrow(/*dim=*/0, /*start=*/0, /*length=*/num_to_keep).to(
                         order_t.device(), keep.scalar_type())
                     }).sort(0, false));
}

template <typename T>
__global__ void hash_rects_kernel(int64_t nthreads,
        T* dets,
        T w0, T h0, T alpha,
        T bx, T by,
        T alpha_ratio,
        int64_t* out) {
    CUDA_1D_KERNEL_LOOP(idx_box, nthreads) {
        auto log_w0 = log(w0);
        auto log_h0 = log(h0);
        auto log_alpha = log(alpha);

        auto curr_det = dets + idx_box * 4;
        auto x = curr_det[0];
        auto y = curr_det[1];
        auto w = curr_det[2];
        auto h = curr_det[3];
        auto w0_alpha = w0 * alpha_ratio;
        auto h0_alpha = h0 * alpha_ratio;

        auto i = round((log_w0 - log(w)) / log_alpha);
        auto j = round((log_h0 - log(h)) / log_alpha);
        auto di = w0_alpha / pow(alpha, i);
        auto dj = h0_alpha / pow(alpha, j);

        int64_t qx, qy;
        qx = round(x / di - bx);
        qy = round(y / dj - by);
        auto curr_out  = out + 4 * idx_box;
        curr_out[0] = qx;
        curr_out[1] = qy;
        curr_out[2] = i;
        curr_out[3] = j;
    }
}

at::Tensor hash_rects_cuda(const at::Tensor& dets,
               float w0,
               float h0,
               float alpha,
               float bx,
               float by) {
    auto num_box = dets.size(0);
    auto alpha_ratio = (1. - alpha) / (1. + alpha);

    auto result = at::zeros({long(num_box), 4},
            dets.options().dtype(at::kLong));

    AT_DISPATCH_FLOATING_TYPES(dets.type(), "HASH_RECTS", [&] {
            hash_rects_kernel<scalar_t><<<GET_BLOCKS(num_box), CUDA_NUM_THREADS>>>(num_box,
                    dets.data<scalar_t>(),
                    (scalar_t)w0, (scalar_t)h0, (scalar_t)alpha,
                    (scalar_t)bx, (scalar_t)by,
                    alpha_ratio,
                    result.data<int64_t>());
            });
    return result;
}

__global__ void map_code(int num_box,
        int64_t* codes,
        int64_t* codes_as_one) {
    CUDA_1D_KERNEL_LOOP(idx_box, num_box) {
        auto curr_code = codes + 4 * idx_box;
        auto curr_mapped = codes_as_one + idx_box;
        *curr_mapped = curr_code[0] +
            curr_code[1] * 10000 +
            curr_code[2] * 100000000 +
            curr_code[3] * 1000000000000;
    }
}

at::Tensor get_best_idx_each_code(
        at::Tensor codes,
        const at::Tensor& scores) {
    auto num_box = codes.size(0);
    auto codes_as_one = at::zeros({long(num_box)},
            codes.options().dtype(at::kLong));
    map_code<<<GET_BLOCKS(num_box), CUDA_NUM_THREADS>>>(num_box,
            codes.data<int64_t>(),
            codes_as_one.data<int64_t>());
    THCudaCheck(hipGetLastError());

    auto unique_result = at::unique_dim(codes_as_one, 0, // dim
            false, true);

    at::Tensor reverse_index = std::get<1>(unique_result);
    auto count = std::get<0>(unique_result).size(0);

    auto result = at::zeros({long(count)},
            codes.options().dtype(at::kLong));

    // get the maximum confidence score for each code with the atomic operation
    // of atomicMax.
    AT_DISPATCH_FLOATING_TYPES(scores.type(), "HNMS_MAX_IDX_KERNEL", [&] {
        hnms_max_conf_kernel<scalar_t><<<GET_BLOCKS(num_box), CUDA_NUM_THREADS>>>(
                num_box,
                scores.data<scalar_t>(),
                reverse_index.data<int64_t>(),
                result.data<int64_t>());
            });
    THCudaCheck(hipGetLastError());

    AT_DISPATCH_FLOATING_TYPES(scores.type(), "HNMS_MAX_IDX_KERNEL", [&] {
            hnms_max_idx_kernel<scalar_t><<<GET_BLOCKS(num_box), CUDA_NUM_THREADS>>>(
                    num_box,
                    scores.data<scalar_t>(),
                    reverse_index.data_ptr<int64_t>(),
                    result.data<int64_t>());
            // NULL,
            });
    return result;
}

at::Tensor hnms_cuda(const at::Tensor& dets,
               const at::Tensor& scores,
               float w0,
               float h0,
               float alpha,
               float bx,
               float by
               ) {
    AT_ASSERTM(dets.type().is_cuda(), "dets must be a CUDA tensor");
    AT_ASSERTM(scores.type().is_cuda(), "scores must be a CUDA tensor");
    AT_ASSERTM(dets.type() == scores.type(), "dets should have the same type as scores");
    if (dets.numel() == 0) {
      return at::empty({0}, dets.options().dtype(at::kLong).device(at::kCPU));
    }

    auto codes = hash_rects_cuda(dets, w0, h0, alpha, bx, by);
    auto result = get_best_idx_each_code(codes, scores);
    return result;
}

